#include "hip/hip_runtime.h"
%%writefile softmax.cu

#include <iostream>
#include <cmath>
#include <limits>
#include <chrono>
#include <hip/hip_runtime.h>
#include <cfloat>

#define ROWS 100 
#define COLS 200  


__global__ void softmax(float* input, float* output, int M, int N)
{
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int col = threadIdx.x + blockDim.x * blockIdx.x;

    if (row < M && col < N) 
    {
        // Step 1: Find maxVal per row : (repreats for every thread need to optimize)
        float maxVal =  input[row*N];
        for (int j = 0; j < N; j++) 
        {
            maxVal = fmaxf(maxVal, input[row * N + j]);
        }

        // Step 2: Compute denominator
        float denominator = 0.0f;
        for (int j = 0; j < N; j++)
        {
            denominator += expf(input[row * N + j] - maxVal);
        }

        // Step 3: Compute softmax output
        output[row * N + col] = expf(input[row * N + col] - maxVal) / denominator;
    }
}





void softmax_cpu(float* input,float* output,int M,int N)
{
    // input is MxN
    float denominator;
    float maxVal;
    //for each m vector
    for(int i=0;i<M;i++)
    {
        denominator = 0.0f;
        maxVal = input[i*N];

        for (int j = 0; j < N; j++) {
            maxVal = fmaxf(maxVal, input[i * N + j]);
        }


        for(int j=0;j<N;j++)
        {
            denominator += expf(input[i*N + j] - maxVal);
        }


        for(int j=0;j<N;j++)
        {
            output[i*N + j] = expf(input[i*N + j] - maxVal)/denominator;
        }

    }

}

__global__ void softmax_sh(float* input, float* output, int M, int N)
{
    //each block for one row
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int col = threadIdx.x + blockDim.x * blockIdx.x;

    __shared__ float denominator;
    __shared__ float maxVal;

    

    if (row < M && col < N) 
    {
        
        if(threadIdx.x == 0)
        {
          denominator = 0.0f;
          maxVal = -FLT_MAX;
        }

        if(threadIdx.x == 0)
        {
            for (int j = 0; j < N; j++) 
            {
                maxVal = fmaxf(maxVal, input[row * N + j]);
            }

        }
        __syncthreads();
        
        // Step 2: Compute denominator
        if(threadIdx.x == 0)
        {
            for (int j = 0; j < N; j++)
            {
                denominator += expf(input[row * N + j] - maxVal);
            }

        }
        
        __syncthreads();
        // Step 3: Compute softmax output
        output[row * N + col] = expf(input[row * N + col] - maxVal) / denominator;
    }
}



int main()
{
    float input[ROWS * COLS], output_cpu[ROWS * COLS], output_gpu[ROWS * COLS];

    for (int i = 0; i < ROWS * COLS; i++)
    {
        input[i] = static_cast<float>(rand() % 10) / 10.0f;
    }

    auto start_cpu = std::chrono::high_resolution_clock::now();
    softmax_cpu(input, output_cpu, ROWS, COLS);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> cpu_time = end_cpu - start_cpu;
    std::cout << "CPU Time: " << cpu_time.count() << " ms" << std::endl;

    float *d_input, *d_output;
    hipMalloc(&d_input,ROWS * COLS* sizeof(float));
    hipMalloc(&d_output, ROWS * COLS * sizeof(float));

    hipMemcpy(d_input, input, ROWS * COLS * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockDim(COLS);
    dim3 gridDim(1, ROWS);


    hipEvent_t start_gpu, stop_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    hipEventRecord(start_gpu);
    softmax_sh<<<gridDim, blockDim>>>(d_input, d_output, ROWS, COLS);
    hipEventRecord(stop_gpu);

    hipDeviceSynchronize();
    hipMemcpy(output_gpu, d_output, ROWS * COLS * sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop_gpu);
    float gpu_time;
    hipEventElapsedTime(&gpu_time, start_gpu, stop_gpu);
    std::cout << "GPU Time: " << gpu_time << " ms" << std::endl;

    bool match = true;
    // for (int i = 0; i < ROWS ; i++)
    // {
    //   for(int j=0;j<COLS;j++)
    //   {
    //     std::cout<<output_cpu[i*COLS + j]<<" : "<<output_gpu[i*COLS + j]<<"\n"<<i<<" "<<j<<"\n";
    //   }
        
    // }
    
    for (int i = 0; i < ROWS * COLS; i++)
    {
        if (fabs(output_cpu[i] - output_gpu[i]) > 1e-5) 
        {
            match = false;
            break;
        }
    }

    std::cout << (match ? "Results match!" : "Mismatch in results!") << std::endl;

    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start_gpu);
    hipEventDestroy(stop_gpu);

    return 0;
}
