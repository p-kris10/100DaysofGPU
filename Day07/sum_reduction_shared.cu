#include "hip/hip_runtime.h"
%%writefile sum_reduction_sh.cu
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <chrono>

__global__ void reduce_sh(float *d_in, float *d_out, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;

    extern __shared__ float sdata[];

    if (idx >= N) return;

    sdata[tid] = d_in[idx];
    __syncthreads();

    // Reduction within block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s]; 
        }
        __syncthreads();
    }

    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];  // Store the reduced sum of the block in global memory
    }
}

void reduce_cpu(float *h_in, float *h_out, int N) {
    h_out[0] = 0;
    for (int i = 0; i < N; i++) {
        h_out[0] += h_in[i];
    }
}

int main() {
    const int N = 1 << 24;
    size_t size = N * sizeof(float);

    // initialization
    float *h_in = (float *)malloc(size);
   for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            h_in[i * cols + j] = static_cast<float>(rand()) / RAND_MAX;
        }
    }

    size_t num_threads = 1024;
    int num_blocks = (N + num_threads - 1) / num_threads;

    // Allocate host output memory for intermediate results
    float *h_out = (float *)malloc((num_blocks + 1) * sizeof(float));

    // Device memory allocation
    float *d_in, *d_out;
    hipMalloc((void **)&d_in, size);
    hipMalloc((void **)&d_out, (num_blocks + 1) * sizeof(float));

    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // First pass: reduce within blocks
    reduce_sh<<<num_blocks, num_threads, num_threads * sizeof(float)>>>(d_in, d_out, N);
    hipDeviceSynchronize();

    // Copy the result from device to host (for the reduction across blocks)
    hipMemcpy(h_out, d_out, num_blocks * sizeof(float), hipMemcpyDeviceToHost);

    // Summing the results of each block to get the final GPU sum
    float gpu_final_sum = 0;
    for (int i = 0; i < num_blocks; i++) {
        gpu_final_sum += h_out[i];
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float gpu_time;
    hipEventElapsedTime(&gpu_time, start, stop);

    // CPU summation for comparison
    auto cpu_start = std::chrono::high_resolution_clock::now();
    reduce_cpu(h_in, h_out, N);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cpu_time = cpu_end - cpu_start;

    float cpu_sum = h_out[0];

    // Output results
    printf("GPU Sum: %.2f\n", gpu_final_sum);
    printf("CPU Sum: %.2f\n", cpu_sum);

    if (gpu_final_sum == cpu_sum) {
        printf("Success! GPU and CPU sums match: %.2f\n", gpu_final_sum);
    } else {
        printf("SUM mismatch! GPU: %.2f, CPU: %.2f\n", gpu_final_sum, cpu_sum);
    }

    printf("GPU Time: %.3f ms\n", gpu_time);
    printf("CPU Time: %.3f ms\n", cpu_time.count());

    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    free(h_out);

    return 0;
}
