#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <chrono>

__global__ void reduce_sh(float *d_in, float *d_out, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;

    extern __shared__ float sdata[];

    if (idx >= N) return;

    sdata[tid] = d_in[idx];
    __syncthreads();

    // Reduction within block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];  // Correct indexing: use tid, not idx
        }
        __syncthreads();
    }

    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];  // Store the reduced sum of the block in global memory
    }
}

__global__ void reduce_global(float *d_in, float *d_out, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;

    if (idx >= N) return;

    // Global reduction across blocks
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            d_in[idx] += d_in[idx + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        d_out[blockIdx.x] = d_in[idx];
    }
}

void reduce_cpu(float *h_in, float *h_out, int N) {
    h_out[0] = 0;
    for (int i = 0; i < N; i++) {
        h_out[0] += h_in[i];
    }
}

int main() {
    const int N = 1 << 18;
    size_t size = N * sizeof(float);

    float *h_in = (float *)malloc(size);
    for (int i = 0; i < N; i++) {
        h_in[i] = 1.0f;  // Initialize for known sum
    }

    size_t num_threads = 1024;
    int num_blocks = (N + num_threads - 1) / num_threads;

    float *h_out = (float *)malloc((num_blocks + 1) * sizeof(float));

    float *d_in, *d_out;
    hipMalloc((void **)&d_in, size);
    hipMalloc((void **)&d_out, (num_blocks + 1) * sizeof(float));

    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // First pass: reduce within blocks
    reduce_sh<<<num_blocks, num_threads, num_threads * sizeof(float)>>>(d_in, d_out, N);
    //cudaMemcpy(h_out, d_out, num_blocks * sizeof(float), cudaMemcpyDeviceToHost);

    // Second pass: reduce across blocks
    reduce_sh<<<1, num_blocks, num_blocks * sizeof(float)>>>(d_out, d_out, num_blocks);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float gpu_time;
    hipEventElapsedTime(&gpu_time, start, stop);

    hipMemcpy(h_out, d_out, sizeof(float), hipMemcpyDeviceToHost);
    float gpu_sum = h_out[0];

    auto cpu_start = std::chrono::high_resolution_clock::now();
    reduce_cpu(h_in, h_out, N);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cpu_time = cpu_end - cpu_start;

    float cpu_sum = h_out[0];

    if (gpu_sum == cpu_sum) {
        printf("Success! GPU and CPU sums match: %.2f\n", gpu_sum);
    } else {
        printf("SUM mismatch! GPU: %.2f, CPU: %.2f\n", gpu_sum, cpu_sum);
    }

    printf("GPU Time: %.3f ms\n", gpu_time);
    printf("CPU Time: %.3f ms\n", cpu_time.count());

    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    free(h_out);

    return 0;
}
